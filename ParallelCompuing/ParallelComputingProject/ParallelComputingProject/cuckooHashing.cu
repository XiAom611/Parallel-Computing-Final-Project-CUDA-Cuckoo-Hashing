#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <time.h>
#include <math.h>



unsigned myrand() {
	unsigned a = rand()<<10;
	unsigned b = rand();
	return a + b;
}

__global__ void cuckooHash(
	unsigned* hashTable,
	unsigned* a, unsigned* b,
	unsigned* entry,
	unsigned* function,
	unsigned* collision,
	unsigned n_function, unsigned n, unsigned p)
{
	unsigned k = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned num = function[k];
	unsigned hashValue = ((a[num] * entry[k] + b[num]) % p) % n;
	if (collision[k] == 1 || hashTable[hashValue] == 0xffffffff) {
		hashTable[hashValue] = entry[k];
		function[k] = (num + 1) % n_function;
	}
}

__global__ void detectCollision(
	unsigned* hashTable,
	unsigned* a, unsigned* b,
	unsigned* entry,
	unsigned* function,
	unsigned* collision,
	unsigned n_function, unsigned n, unsigned p)
{
	unsigned k = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned num = (function[k] - 1) % n_function;
	unsigned hashValue = ((a[num] * entry[k] + b[num]) % p) % n;
	if (hashTable[hashValue] != entry[k]) {
		collision[k] = 1;
	} else {
		collision[k] = 0;
	}
}

__global__ void lookup(
	unsigned* hashTable,
	unsigned* a, unsigned* b,
	unsigned* searchEntry,
	unsigned* dict,
	unsigned n_function, unsigned n, unsigned p)
{
	unsigned k = blockDim.x * blockIdx.x + threadIdx.x;
	for (unsigned i = 0; i < n_function; i++) {
		unsigned hashValue = ((a[i] * searchEntry[k] + b[i]) % p) % n;
		if (hashTable[hashValue] == searchEntry[k]) {
			dict[k] = 1;
			break;
		}
	}
}

void generate_a_b(unsigned n_function, unsigned* a, unsigned* b) {
	for (unsigned i = 0; i < n_function; i++) {
		a[i] = rand() % 10;
		b[i] = rand() % 10;
		if (i != 0) {
			while (a[i] == a[i - 1] || b[i] == b[i - 1]) {
				a[i] = rand() % 10;
				b[i] = rand() % 10;
			}
		}
	}

	/////////////For task 5
	//a[0] = 232;
	//b[0] = 0;

	//////////DEBUG
	std::cout << "a: ";
	for (unsigned i = 0; i < n_function; i++) {
		std::cout << a[i] << " ";
	}
	std::cout << std::endl;
	std::cout << "b: ";
	for (unsigned i = 0; i < n_function; i++) {
		std::cout << b[i] << " ";
	}
	std::cout << std::endl;
}

int main() {
	//std::cout << myrand() << std::endl;
	//std::cout << myrand() << std::endl;
	//std::cout << myrand() << std::endl;
	//std::cout << myrand() << std::endl;
	//std::cout << myrand() << std::endl;
	//std::cout << myrand() << std::endl;
	//std::cout << myrand() << std::endl;
	//std::cout << myrand() << std::endl;
	//std::cout << myrand() << std::endl;

	hipError_t err = hipSuccess;

	unsigned task, n_function;
	unsigned N;
	unsigned entryLength;
	unsigned p = 62353171; // a large prime

	unsigned *entry;
	unsigned *collision;
	unsigned *hashTable;
	unsigned *function;
	unsigned *a, *b;
	unsigned *dict;
	unsigned *searchEntry;

	unsigned *d_entry = NULL;
	unsigned *d_collision = NULL;
	unsigned *d_hashTable = NULL;
	unsigned *d_function = NULL;
	unsigned *d_a = NULL;
	unsigned *d_b = NULL;
	unsigned *d_dict = NULL;
	unsigned *d_searchEntry = NULL;

	unsigned limit;
	unsigned blockNum;
	unsigned blockSize = 512;
	unsigned flag = 0;

	unsigned iteration = 0;

	unsigned test;
	unsigned testNum;
	unsigned testHashValue;

	clock_t startTime;
	clock_t endTime;

	unsigned sum = 0;

	std::cout << "How many hash functions?" << std::endl;
	std::cin >> n_function;
	std::cout << "Which task?" << std::endl;
	std::cin >> task;



	srand(time(NULL));

	switch (task)
	{
	case 1:
		N = pow(2, 25); //33554432
		limit = ceil(4 * log10((double)N));
		unsigned s;
		std::cout << "Input s:";
		std::cin >> s;
		entryLength = pow(2, s);

		for (unsigned z = 0; z < 5; z++) {
			blockNum = ceil((double)entryLength / blockSize);
			entry = new unsigned[entryLength];

			std::cout << "Generating random numbers between 0~10000000..." << std::endl;
			for (unsigned i = 0; i < entryLength; i++) {
				entry[i] = myrand() % 10000000;
			}

			std::cout << "Generating a,b..." << std::endl;
			a = new unsigned[n_function];
			b = new unsigned[n_function];
			generate_a_b(n_function, a, b);
			//for (unsigned i = 0; i < n_function; i++) {
			//	a[i] = rand() % 10;
			//	b[i] = rand() % 10;
			//	if (i != 0) {
			//		while (a[i]==a[i-1] || b[i]==b[i-1]){
			//			a[i] = rand() % 10;
			//			b[i] = rand() % 10;
			//		}
			//	}
			//}

			std::cout << "Initilizing hashTable..." << std::endl;
			hashTable = new unsigned[N];
			memset(hashTable, 0xffffffff, N * sizeof(unsigned));

			std::cout << "Initilizing collisionTable..." << std::endl;
			collision = new unsigned[entryLength];
			memset(collision, 0, entryLength * sizeof(unsigned));

			std::cout << "Initilizing functionIndex..." << std::endl;
			function = new unsigned[entryLength];
			memset(function, 0, entryLength * sizeof(unsigned));

			std::cout << "Allocating device memory..." << std::endl;

			err = hipMalloc((void**)&d_entry, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating entry[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_a, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating a[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_b, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating b[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_collision, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating collision[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_function, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating functionIndex[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_hashTable, N * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating hashTable[] failed" << std::endl;
				goto Error;
			}

			std::cout << "Copying memory from host to device..." << std::endl;

			err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy hashTable" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy a[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy b[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy entry[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy functionIndex[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy collision[]" << std::endl;
				goto Error;
			}
			iteration = 0;
			startTime = clock();
			do {
				flag = 0;
				//Restarting hash
				if (iteration == limit) {
					iteration = 0;
					std::cout << ".........Rehash........." << std::endl;
					generate_a_b(n_function, a, b);
					memset(hashTable, 0xffffffff, N * sizeof(unsigned));
					memset(function, 0, entryLength * sizeof(unsigned));
					memset(collision, 0, entryLength * sizeof(unsigned));

					std::cout << "Recopying memory from host to device..." << std::endl;

					err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy hashTable" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy a[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy b[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy entry[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy functionIndex[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy collision[]" << std::endl;
						goto Error;
					}
				}



				iteration++;
				cuckooHash << < blockNum, blockSize >> > (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				detectCollision << < blockNum, blockSize >> > (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				std::cout << "Finish hash " << iteration << " times" << std::endl;

				// Copy collison back
				err = hipMemcpy(collision, d_collision, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
				if (err != hipSuccess) {
					std::cout << "Copy collison failed from device to host" << std::endl;
					std::cout << hipGetErrorString(err) << std::endl;
					goto Error;
				}

				for (unsigned i = 0; i < entryLength; i++) {
					flag += collision[i];
				}
				std::cout << flag << " collisions" << std::endl;

			} while (flag != 0);
			endTime = clock();
			std::cout << "Hash Done!" << std::endl;

			std::cout << "time:" << (double)(endTime - startTime) / CLOCKS_PER_SEC << "s" << std::endl;
			std::cout << std::endl;

			//////////test
			//hipMemcpy(function, d_function, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
			//hipMemcpy(hashTable, d_hashTable, N * sizeof(unsigned), hipMemcpyDeviceToHost);
			//std::cout << "Checking Hash..." << std::endl;
			//for (unsigned i = 0; i < 10; i++) {
			//	test = myrand() % entryLength;
			//	testNum = (function[test] - 1) % n_function;
			//	testHashValue = ((a[testNum] * entry[test] + b[testNum]) % p) % N;
			//	std::cout << "--" << entry[test] << " " << testNum << " ";
			//	if (hashTable[testHashValue] == entry[test]) {
			//		std::cout << "correct" << std::endl;
			//	} else {
			//		std::cout << "incorrect" << std::endl;
			//	}
			//}
		}
		break;

	case 2:
		N = pow(2, 25); //33554432
		limit = ceil(4 * log10((double)N));
		unsigned ii;
		std::cout << "Input i:";
		std::cin >> ii;
		entryLength = pow(2, 24);

		for (unsigned z = 0; z < 5; z++) {
			blockNum = ceil((double)entryLength / blockSize);
			entry = new unsigned[entryLength];

			std::cout << "Generating random numbers between 0~10000000..." << std::endl;
			for (unsigned i = 0; i < entryLength; i++) {
				entry[i] = myrand() % 10000000;
			}

			std::cout << "Generating a,b..." << std::endl;
			a = new unsigned[n_function];
			b = new unsigned[n_function];
			generate_a_b(n_function, a, b);
			//for (unsigned i = 0; i < n_function; i++) {
			//	a[i] = rand() % 10;
			//	b[i] = rand() % 10;
			//	if (i != 0) {
			//		while (a[i]==a[i-1] || b[i]==b[i-1]){
			//			a[i] = rand() % 10;
			//			b[i] = rand() % 10;
			//		}
			//	}
			//}

			std::cout << "Initilizing hashTable..." << std::endl;
			hashTable = new unsigned[N];
			memset(hashTable, 0xffffffff, N * sizeof(unsigned));

			std::cout << "Initilizing collisionTable..." << std::endl;
			collision = new unsigned[entryLength];
			memset(collision, 0, entryLength * sizeof(unsigned));

			std::cout << "Initilizing functionIndex..." << std::endl;
			function = new unsigned[entryLength];
			memset(function, 0, entryLength * sizeof(unsigned));

			std::cout << "Allocating device memory..." << std::endl;

			err = hipMalloc((void**)&d_entry, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating entry[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_a, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating a[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_b, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating b[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_collision, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating collision[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_function, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating functionIndex[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_hashTable, N * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating hashTable[] failed" << std::endl;
				goto Error;
			}

			std::cout << "Copying memory from host to device..." << std::endl;

			err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy hashTable" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy a[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy b[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy entry[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy functionIndex[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy collision[]" << std::endl;
				goto Error;
			}
			iteration = 0;
			do {
				flag = 0;
				//Restarting hash
				if (iteration == limit) {
					iteration = 0;
					std::cout << ".........Rehash........." << std::endl;
					generate_a_b(n_function, a, b);
					memset(hashTable, 0xffffffff, N * sizeof(unsigned));
					memset(function, 0, entryLength * sizeof(unsigned));
					memset(collision, 0, entryLength * sizeof(unsigned));

					std::cout << "Recopying memory from host to device..." << std::endl;

					err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy hashTable" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy a[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy b[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy entry[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy functionIndex[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy collision[]" << std::endl;
						goto Error;
					}
				}

				iteration++;
				cuckooHash <<< blockNum, blockSize >>> (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				detectCollision <<< blockNum, blockSize >>> (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				std::cout << "Finish hash " << iteration << " times" << std::endl;

				// Copy collison back
				err = hipMemcpy(collision, d_collision, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
				if (err != hipSuccess) {
					std::cout << "Copy collison failed from device to host" << std::endl;
					goto Error;
				}

				for (unsigned i = 0; i < entryLength; i++) {
					flag += collision[i];
				}
				std::cout << flag << " collisions" << std::endl;

			} while (flag != 0);
			std::cout << "Hash Done!" << std::endl;

			//////////////////////////////////
			///////// Look up part
			std::cout << "Initilizing lookup part..." << std::endl;

			hipMemcpy(hashTable, d_hashTable, N * sizeof(unsigned), hipMemcpyDeviceToHost);

			searchEntry = new unsigned[entryLength];
			dict = new unsigned[entryLength];

			//add random key in to search entry
			for (unsigned i = 0; i < entryLength; i++) {
				//std::cout << myrand() << std::endl;
				if (i < entryLength*(1 - ii*0.1)) {
					unsigned randIdx = myrand() % entryLength;
					//std::cout << randIdx << " ";
					searchEntry[i] = entry[randIdx];
				} else {
					searchEntry[i] = myrand();
				}
			}
			//std::cout << std::endl;

			//store if find
			memset(dict, 0, entryLength * sizeof(unsigned));

			//allocate cuda memory for search
			err = hipMalloc((void**)&d_searchEntry, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "allocate searchEntry fail" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_dict, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "allocate dict fail" << std::endl;
				goto Error;
			}

			//copy data from host to device
			err = hipMemcpy(d_searchEntry, searchEntry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "Fail to copy search Entry to device" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_dict, dict, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "Fail to copy dict to device" << std::endl;
				goto Error;
			}

			std::cout << "Start lookup keys..." << std::endl;

			///////////////DEBUG
			//sum = 0;
			//for (unsigned i = 0; i < entryLength; i++) {
			//	for (unsigned j = 0; j < n_function; j++) {
			//		unsigned hashValue = ((a[j] * searchEntry[i] + b[j]) % p) % N;
			//		if (hashTable[hashValue] == searchEntry[i]) {
			//			sum++;
			//			break;
			//		}
			//	}
			//}
			//std::cout << "--------------" << sum/entryLength << std::endl;

			startTime = clock();
			lookup <<< blockNum, blockSize >>> (d_hashTable,
				d_a, d_b,
				d_searchEntry,
				d_dict,
				n_function, N, p);
			endTime = clock();
			std::cout << "Lookup done" << std::endl;

			//Copy dict back to host
			err = hipMemcpy(dict, d_dict, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
			if (err != hipSuccess) {
				std::cout << "Fail to copy dict back to device" << std::endl;
				goto Error;
			}

			sum = 0;
			for (unsigned i = 0; i < entryLength; i++) {
				sum += dict[i];
			}

			std::cout << "Hash Hit: " << (double)sum / (double)entryLength * 100 << "%";
			std::cout << " with " << (double)(endTime - startTime) / CLOCKS_PER_SEC << "s" << std::endl;
			std::cout << std::endl;
		}
		break;

	case 3:
		//N = pow(2, 25); //33554432
		float alpha;
		std::cout << "input alpha: ";
		std::cin >> alpha;

		entryLength = pow(2, 24);
		std::cout << entryLength << " ";

		N = alpha*entryLength;
		std::cout << N << " ";
		limit = ceil(4 * log10((double)N));
		std::cout << limit << std::endl;


		for (unsigned z = 0; z < 5; z++) {
			blockNum = ceil((double)entryLength / blockSize);
			entry = new unsigned[entryLength];

			std::cout << "Generating random numbers between 0~10000000..." << std::endl;
			for (unsigned i = 0; i < entryLength; i++) {
				entry[i] = myrand() % 10000000;
			}

			std::cout << "Generating a,b..." << std::endl;
			a = new unsigned[n_function];
			b = new unsigned[n_function];
			generate_a_b(n_function, a, b);
			//for (unsigned i = 0; i < n_function; i++) {
			//	a[i] = rand() % 10;
			//	b[i] = rand() % 10;
			//	if (i != 0) {
			//		while (a[i]==a[i-1] || b[i]==b[i-1]){
			//			a[i] = rand() % 10;
			//			b[i] = rand() % 10;
			//		}
			//	}
			//}

			std::cout << "Initilizing hashTable..." << std::endl;
			hashTable = new unsigned[N];
			memset(hashTable, 0xffffffff, N * sizeof(unsigned));

			std::cout << "Initilizing collisionTable..." << std::endl;
			collision = new unsigned[entryLength];
			memset(collision, 0, entryLength * sizeof(unsigned));

			std::cout << "Initilizing functionIndex..." << std::endl;
			function = new unsigned[entryLength];
			memset(function, 0, entryLength * sizeof(unsigned));

			std::cout << "Allocating device memory..." << std::endl;

			err = hipMalloc((void**)&d_entry, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating entry[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_a, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating a[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_b, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating b[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_collision, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating collision[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_function, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating functionIndex[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_hashTable, N * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating hashTable[] failed" << std::endl;
				goto Error;
			}

			std::cout << "Copying memory from host to device..." << std::endl;

			err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy hashTable" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy a[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy b[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy entry[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy functionIndex[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy collision[]" << std::endl;
				goto Error;
			}

			iteration = 0;
			startTime = clock();
			do {
				flag = 0;
				//Restarting hash
				if (iteration == limit) {
					iteration = 0;
					std::cout << ".........Rehash........." << std::endl;
					generate_a_b(n_function, a, b);
					memset(hashTable, 0xffffffff, N * sizeof(unsigned));
					memset(function, 0, entryLength * sizeof(unsigned));
					memset(collision, 0, entryLength * sizeof(unsigned));

					std::cout << "Recopying memory from host to device..." << std::endl;

					err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy hashTable" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy a[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy b[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy entry[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy functionIndex[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy collision[]" << std::endl;
						goto Error;
					}
				}



				iteration++;
				cuckooHash << < blockNum, blockSize >> > (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				detectCollision << < blockNum, blockSize >> > (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				std::cout << "Finish hash " << iteration << " times" << std::endl;

				// Copy collison back
				err = hipMemcpy(collision, d_collision, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
				if (err != hipSuccess) {
					std::cout << "Copy collison failed from device to host" << std::endl;
					std::cout << hipGetErrorString(err) << std::endl;
					goto Error;
				}

				for (unsigned i = 0; i < entryLength; i++) {
					flag += collision[i];
				}
				std::cout << flag << " collisions" << std::endl;

			} while (flag != 0);
			endTime = clock();
			std::cout << "Hash Done!" << std::endl;

			std::cout << "time:" << (double)(endTime - startTime) / CLOCKS_PER_SEC << "s" << std::endl;
			std::cout << std::endl;

			////////////test
			//hipMemcpy(function, d_function, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
			//hipMemcpy(hashTable, d_hashTable, N * sizeof(unsigned), hipMemcpyDeviceToHost);
			//std::cout << "Checking Hash..." << std::endl;
			//for (unsigned i = 0; i < 10; i++) {
			//	test = myrand() % entryLength;
			//	testNum = (function[test] - 1) % n_function;
			//	testHashValue = ((a[testNum] * entry[test] + b[testNum]) % p) % N;
			//	std::cout << "--" << entry[test] << " " << testNum << " ";
			//	if (hashTable[testHashValue] == entry[test]) {
			//		std::cout << "correct" << std::endl;
			//	} else {
			//		std::cout << "incorrect" << std::endl;
			//	}
			//}
		}
		break;

	case 4:
		float beta;
		std::cout << "input bound coefficient: ";
		std::cin >> beta;

		entryLength = pow(2, 24);
		std::cout << entryLength << " ";

		N = 1.2*entryLength;
		std::cout << N << " ";
		limit = ceil(beta * log10((double)N));
		std::cout << limit << std::endl;


		for (unsigned z = 0; z < 5; z++) {
			blockNum = ceil((double)entryLength / blockSize);
			entry = new unsigned[entryLength];

			std::cout << "Generating random numbers between 0~10000000..." << std::endl;
			for (unsigned i = 0; i < entryLength; i++) {
				entry[i] = myrand() % 10000000;
			}

			std::cout << "Generating a,b..." << std::endl;
			a = new unsigned[n_function];
			b = new unsigned[n_function];
			generate_a_b(n_function, a, b);
			//for (unsigned i = 0; i < n_function; i++) {
			//	a[i] = rand() % 10;
			//	b[i] = rand() % 10;
			//	if (i != 0) {
			//		while (a[i]==a[i-1] || b[i]==b[i-1]){
			//			a[i] = rand() % 10;
			//			b[i] = rand() % 10;
			//		}
			//	}
			//}

			std::cout << "Initilizing hashTable..." << std::endl;
			hashTable = new unsigned[N];
			memset(hashTable, 0xffffffff, N * sizeof(unsigned));

			std::cout << "Initilizing collisionTable..." << std::endl;
			collision = new unsigned[entryLength];
			memset(collision, 0, entryLength * sizeof(unsigned));

			std::cout << "Initilizing functionIndex..." << std::endl;
			function = new unsigned[entryLength];
			memset(function, 0, entryLength * sizeof(unsigned));

			std::cout << "Allocating device memory..." << std::endl;

			err = hipMalloc((void**)&d_entry, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating entry[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_a, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating a[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_b, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating b[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_collision, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating collision[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_function, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating functionIndex[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_hashTable, N * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating hashTable[] failed" << std::endl;
				goto Error;
			}

			std::cout << "Copying memory from host to device..." << std::endl;

			err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy hashTable" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy a[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy b[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy entry[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy functionIndex[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy collision[]" << std::endl;
				goto Error;
			}

			iteration = 0;
			startTime = clock();
			do {
				flag = 0;
				//Restarting hash
				if (iteration == limit) {
					iteration = 0;
					std::cout << ".........Rehash........." << std::endl;
					generate_a_b(n_function, a, b);
					memset(hashTable, 0xffffffff, N * sizeof(unsigned));
					memset(function, 0, entryLength * sizeof(unsigned));
					memset(collision, 0, entryLength * sizeof(unsigned));

					std::cout << "Recopying memory from host to device..." << std::endl;

					err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy hashTable" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy a[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy b[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy entry[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy functionIndex[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy collision[]" << std::endl;
						goto Error;
					}
				}



				iteration++;
				cuckooHash << < blockNum, blockSize >> > (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				detectCollision << < blockNum, blockSize >> > (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				std::cout << "Finish hash " << iteration << " times" << std::endl;

				// Copy collison back
				err = hipMemcpy(collision, d_collision, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
				if (err != hipSuccess) {
					std::cout << "Copy collison failed from device to host" << std::endl;
					std::cout << hipGetErrorString(err) << std::endl;
					goto Error;
				}

				for (unsigned i = 0; i < entryLength; i++) {
					flag += collision[i];
				}
				std::cout << flag << " collisions" << std::endl;

			} while (flag != 0);
			endTime = clock();
			std::cout << "Hash Done!" << std::endl;

			std::cout << "time:" << (double)(endTime - startTime) / CLOCKS_PER_SEC << "s" << std::endl;
			std::cout << std::endl;

			//////////test
			//hipMemcpy(function, d_function, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
			//hipMemcpy(hashTable, d_hashTable, N * sizeof(unsigned), hipMemcpyDeviceToHost);
			//std::cout << "Checking Hash..." << std::endl;
			//for (unsigned i = 0; i < 10; i++) {
			//	test = myrand() % entryLength;
			//	testNum = (function[test] - 1) % n_function;
			//	testHashValue = ((a[testNum] * entry[test] + b[testNum]) % p) % N;
			//	std::cout << "--" << entry[test] << " " << testNum << " ";
			//	if (hashTable[testHashValue] == entry[test]) {
			//		std::cout << "correct" << std::endl;
			//	} else {
			//		std::cout << "incorrect" << std::endl;
			//	}
			//}
		}
		break;
	case 5:
		entryLength = pow(2, 24);
		std::cout << entryLength << " ";

		N = 1.2*entryLength;
		std::cout << N << " ";
		limit = ceil(6 * log10((double)N));
		std::cout << limit << std::endl;


		for (unsigned z = 0; z < 5; z++) {
			blockNum = ceil((double)entryLength / blockSize);
			entry = new unsigned[entryLength];

			std::cout << "Generating random numbers between 0~10000000..." << std::endl;
			for (unsigned i = 0; i < entryLength; i++) {
				entry[i] = myrand() % 10000000;
			}

			std::cout << "Generating a,b..." << std::endl;
			a = new unsigned[n_function];
			b = new unsigned[n_function];
			generate_a_b(n_function, a, b);
			//for (unsigned i = 0; i < n_function; i++) {
			//	a[i] = rand() % 10;
			//	b[i] = rand() % 10;
			//	if (i != 0) {
			//		while (a[i]==a[i-1] || b[i]==b[i-1]){
			//			a[i] = rand() % 10;
			//			b[i] = rand() % 10;
			//		}
			//	}
			//}

			std::cout << "Initilizing hashTable..." << std::endl;
			hashTable = new unsigned[N];
			memset(hashTable, 0xffffffff, N * sizeof(unsigned));

			std::cout << "Initilizing collisionTable..." << std::endl;
			collision = new unsigned[entryLength];
			memset(collision, 0, entryLength * sizeof(unsigned));

			std::cout << "Initilizing functionIndex..." << std::endl;
			function = new unsigned[entryLength];
			memset(function, 0, entryLength * sizeof(unsigned));

			std::cout << "Allocating device memory..." << std::endl;

			err = hipMalloc((void**)&d_entry, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating entry[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_a, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating a[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_b, n_function * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating b[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_collision, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating collision[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_function, entryLength * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating functionIndex[] failed" << std::endl;
				goto Error;
			}

			err = hipMalloc((void**)&d_hashTable, N * sizeof(unsigned));
			if (err != hipSuccess) {
				std::cout << "-->Allocating hashTable[] failed" << std::endl;
				goto Error;
			}

			std::cout << "Copying memory from host to device..." << std::endl;

			err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy hashTable" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy a[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy b[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy entry[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy functionIndex[]" << std::endl;
				goto Error;
			}

			err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
			if (err != hipSuccess) {
				std::cout << "-->Fail to copy collision[]" << std::endl;
				goto Error;
			}

			iteration = 0;
			startTime = clock();
			do {
				flag = 0;
				//Restarting hash
				if (iteration == limit) {
					iteration = 0;
					std::cout << ".........Rehash........." << std::endl;
					generate_a_b(n_function, a, b);
					memset(hashTable, 0xffffffff, N * sizeof(unsigned));
					memset(function, 0, entryLength * sizeof(unsigned));
					memset(collision, 0, entryLength * sizeof(unsigned));

					std::cout << "Recopying memory from host to device..." << std::endl;

					err = hipMemcpy(d_hashTable, hashTable, N * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy hashTable" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_a, a, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy a[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_b, b, n_function * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy b[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_entry, entry, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy entry[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_function, function, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy functionIndex[]" << std::endl;
						goto Error;
					}

					err = hipMemcpy(d_collision, collision, entryLength * sizeof(unsigned), hipMemcpyHostToDevice);
					if (err != hipSuccess) {
						std::cout << "-->Fail to copy collision[]" << std::endl;
						goto Error;
					}
				}



				iteration++;
				cuckooHash << < blockNum, blockSize >> > (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				detectCollision << < blockNum, blockSize >> > (d_hashTable,
					d_a, d_b,
					d_entry,
					d_function,
					d_collision,
					n_function, N, p);

				std::cout << "Finish hash " << iteration << " times" << std::endl;

				// Copy collison back
				err = hipMemcpy(collision, d_collision, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
				if (err != hipSuccess) {
					std::cout << "Copy collison failed from device to host" << std::endl;
					std::cout << hipGetErrorString(err) << std::endl;
					goto Error;
				}

				for (unsigned i = 0; i < entryLength; i++) {
					flag += collision[i];
				}
				std::cout << flag << " collisions" << std::endl;

			} while (flag != 0);
			endTime = clock();
			std::cout << "Hash Done!" << std::endl;

			std::cout << "time:" << (double)(endTime - startTime) / CLOCKS_PER_SEC << "s" << std::endl;
			std::cout << std::endl;

			//////////test
			//hipMemcpy(function, d_function, entryLength * sizeof(unsigned), hipMemcpyDeviceToHost);
			//hipMemcpy(hashTable, d_hashTable, N * sizeof(unsigned), hipMemcpyDeviceToHost);
			//std::cout << "Checking Hash..." << std::endl;
			//for (unsigned i = 0; i < 10; i++) {
			//	test = myrand() % entryLength;
			//	testNum = (function[test] - 1) % n_function;
			//	testHashValue = ((a[testNum] * entry[test] + b[testNum]) % p) % N;
			//	std::cout << "--" << entry[test] << " " << testNum << " ";
			//	if (hashTable[testHashValue] == entry[test]) {
			//		std::cout << "correct" << std::endl;
			//	} else {
			//		std::cout << "incorrect" << std::endl;
			//	}
			//}
		}
		break;

	default:
		std::cout << "No such task. Please run again." << std::endl;
		goto Error;
		break;
	}



Error:

	hipFree((void**)&d_a);
	hipFree((void**)&d_b);
	hipFree((void**)&d_collision);
	hipFree((void**)&d_dict);
	hipFree((void**)&d_entry);
	hipFree((void**)&d_function);
	hipFree((void**)&d_searchEntry);
	hipFree((void**)&d_hashTable);
	system("pause");
	return 0;
}
